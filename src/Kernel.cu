#include "hip/hip_runtime.h"
#include "../include/Kernel.h"
//#include "../include/vector_ops.h"

__host__ __device__ int4 discreteDisAndAngle(float4 ppf, float min_angle, float d_angle) {
    int4 r;

    if(ppf.x == -1) {
        r.x = -1;
        return r;
    }

    r.x = int(ppf.x);
    r.y = int((ppf.y - min_angle) * d_angle);
    r.z = int((ppf.z - min_angle) * d_angle);
    r.w = int((ppf.w - min_angle) * d_angle);
    return r;
}

__host__ __device__ float4 computePPF(float3 p1, float3 n1, float3 p2, float3 n2, float d_dist) {
    /*
    *这里的PPF计算、离散、计算Hash都采用 PPF.cpp 的计算公式，可对照cpu版公式理解涵义
    */
    float3 d;
    d.x = p2.x - p1.x;
    d.y = p2.y - p1.y;
    d.z = p2.z - p1.z;
    float4 f;
    f.x = norm(d) / d_dist;

    // because tau_d is 0.05
    if(f.x > 20) {
        f.x = -1;
        return f;
    }

    d=normalize(d);
    f.y = atan2f(norm(cross(d,n1)), dot(d,n1));;
    f.z = atan2f(norm(cross(d,n2)), dot(d,n2));
    f.w = atan2f(norm(cross(n1,n2)), dot(n1,n2));
    return f;
}

__host__ __device__ unsigned int hash(int4 ppf){
    return (ppf.x * P1 + ppf.y * P2 + ppf.z * P3 + ppf.w * P4) % MAX_HASHINDEX;
}

__host__ __device__ __forceinline__ void zeroMat4(float T[4][4]) {
    T[0][0] = 0;    T[0][1] = 0;    T[0][2] = 0;    T[0][3] = 0;
    T[1][0] = 0;    T[1][1] = 0;    T[1][2] = 0;    T[1][3] = 0;
    T[2][0] = 0;    T[2][1] = 0;    T[2][2] = 0;    T[2][3] = 0;
    T[3][0] = 0;    T[3][1] = 0;    T[3][2] = 0;    T[3][3] = 0;
}

__host__ __device__ void trans(float3 v, float T[4][4]) {
    //构建平移矩阵
    zeroMat4(T);
    T[0][0] = 1;
    T[1][1] = 1;
    T[2][2] = 1;
    T[3][3] = 1;
    T[0][3] = v.x;
    T[1][3] = v.y;
    T[2][3] = v.z;
}

__host__ __device__ void rotx(float theta, float T[4][4]) {
    //向X轴旋转的旋转矩阵
    zeroMat4(T);
    T[0][0] = 1;
    T[1][1] = cosf(theta);
    T[2][1] = sinf(theta);
    T[1][2] = -1*T[2][1];
    T[2][2] = T[1][1];
    T[3][3] = 1;
}

__host__ __device__ void roty(float theta, float T[4][4]) {
    //向Y轴旋转的旋转矩阵
    zeroMat4(T);
    T[0][0] = cosf(theta);
    T[0][2] = sinf(theta);
    T[1][1] = 1;
    T[2][0] = -1*T[0][2];
    T[2][2] = T[0][0];
    T[3][3] = 1;
}

__host__ __device__ void rotz(float theta, float T[4][4]) {
    //向Z轴旋转的旋转矩阵
    zeroMat4(T);
    T[0][0] = cosf(theta);
    T[1][0] = sinf(theta);
    T[0][1] = -1*T[1][0];
    T[1][1] = T[0][0];
    T[2][2] = 1;
    T[3][3] = 1;
}

__host__ __device__ float4 homogenize(float3 v) {
    float4 w = {v.x, v.y, v.z, 1};
    return w;
}

__host__ __device__ float3 dehomogenize(float4 v) {
    float3 w = {v.x, v.y, v.z};
    return w;
}

__host__ __device__ float4 mat4fVmul(const float A[4][4], const float4 b) {
    float4 *Af4 = (float4 *) A;
    float4 c;
    c.x = dot(Af4[0], b);
    c.y = dot(Af4[1], b);
    c.z = dot(Af4[2], b);
    c.w = dot(Af4[3], b);
    return c;
}

__host__ __device__ void mat4fMul(const float A[4][4], const float B[4][4], float C[4][4]) {
    zeroMat4(C);
    for(int i = 0; i < 4; i++){
        for(int j = 0; j < 4; j++){
            for(int k = 0; k < 4; k++){
                C[i][j] += A[i][k]*B[k][j];
            }
        }
    }
}

__host__ __device__ float quantDownf(float x, float y) {
    return x - fmodf(x, y);
}

__device__ void transModelAndScene(float3 m_r, float3 n_r_m, float3 m_i,
                                  float3 s_r, float3 n_r_s, float3 s_i,
                                  float d_dist, unsigned int &alpha_idx){
    /*按照论文3.3节公式，计算转角：
    *用来计算把点对一端旋转到X轴上，另一端的夹角
    *后面不采用这个大函数，拆分为：transPointPair()  和  transPointPair()
    */
    float transm[4][4], rot_x[4][4], rot_y[4][4], rot_z[4][4], T_tmp[4][4], T_m_g[4][4], T_s_g[4][4],
            T_tmp2[4][4], T[4][4];
    float4 n_tmp;
    //旋转model中的点对
    m_r = -1 * m_r;
    trans(m_r, transm);
    roty(atan2f(n_r_m.z, n_r_m.x), rot_y);
    n_tmp = homogenize(n_r_m);
    n_tmp = mat4fVmul(rot_y, n_tmp);
    rotz(-1*atan2f(n_tmp.y, n_tmp.x), rot_z);
    mat4fMul(rot_z, rot_y, T_tmp);
    mat4fMul(T_tmp, transm, T_m_g);
    //旋转scene中的点对
    s_r = -1 * s_r;
    trans(s_r, transm);
    roty(atan2f(n_r_s.z, n_r_s.x), rot_y);
    n_tmp = homogenize(n_r_s);
    n_tmp = mat4fVmul(rot_y, n_tmp);
    rotz(-1*atan2f(n_tmp.y, n_tmp.x), rot_z);
    mat4fMul(rot_z, rot_y, T_tmp);
    mat4fMul(T_tmp, transm, T_s_g);
    //计算model夹角
    n_tmp = homogenize(m_i);
    n_tmp = mat4fVmul(T_m_g, n_tmp);
    float3 u = dehomogenize(n_tmp);
    //计算scene夹角
    n_tmp = homogenize(s_i);
    n_tmp = mat4fVmul(T_s_g, n_tmp);
    float3 v = dehomogenize(n_tmp);
    //计算夹角差值
    u.x = 0;
    v.x = 0;
    float alpha = atan2f(cross(u, v).x, dot(u, v));
    alpha = quantDownf(alpha + HIP_PI_F, D_ANGLE0);
    alpha_idx = (unsigned int) (lrintf(alpha/D_ANGLE0));
    rotx(alpha, rot_x);
}

__device__ void transPointPair(float3 m_r, float3 n_r_m, float3 m_i, float &alpha){
    float transm[4][4], rot_x[4][4], rot_y[4][4], rot_z[4][4], T_tmp[4][4], T_m_g[4][4], T_s_g[4][4], T_tmp2[4][4], T[4][4];
    float4 n_tmp;
    float3 u;
    m_r = -1 * m_r;
    //计算平移矩阵
    trans(m_r, transm);
    //计算绕Y轴旋转的旋转矩阵rot_y
    roty(atan2f(n_r_m.z, n_r_m.x), rot_y);
    n_tmp = homogenize(n_r_m);
    n_tmp = mat4fVmul(rot_y, n_tmp);
    //计算绕Z轴旋转的旋转矩阵rot_z
    rotz(-1*atan2f(n_tmp.y, n_tmp.x), rot_z);
    //combina three transports into one T_m_g
    mat4fMul(rot_z, rot_y, T_tmp);
    mat4fMul(T_tmp, transm, T_m_g);

    n_tmp = homogenize(m_i);
    n_tmp = mat4fVmul(T_m_g, n_tmp);

    u = dehomogenize(n_tmp);
    float rot_angle = atan2f(u.y,u.z);
    alpha = rot_angle;
}

__device__ void transModelScene(float u, float v, unsigned int &alpha_idx) {
    /*float alpha = atan2f(cross(u, v).x, dot(u, v));
    alpha = quantDownf(alpha + HIP_PI_F, D_ANGLE0);
    alpha_idx = (unsigned int) (lrintf(alpha/D_ANGLE0));*/
    float alpha_m2s = u - v;

    if (alpha_m2s < -M_PI)
        alpha_m2s += M_PI * 2;
    if (alpha_m2s > M_PI)
        alpha_m2s -= M_PI * 2;

    alpha_idx = (alpha_m2s - MIN_ANGLE)*D_ANGLE;
}

__device__ bool isFeatureSimilar(int4 scenePPF, int4 modelPPF, float dis_thresh, float angle_thresh) {
    /*
    *因为做了离散处理，两个点对要判等得完全相等
    *这个判等在CUDA十分影响计算时间，增加约一倍！后面可以采取策略消除掉
    */
    if (fabsf(scenePPF.x - modelPPF.x) > dis_thresh || fabsf(scenePPF.y - modelPPF.y) > angle_thresh || 
        fabsf(scenePPF.z - modelPPF.z) > angle_thresh || fabsf(scenePPF.w - modelPPF.w) > angle_thresh)
        return false;
    return true;
}

__device__ bool isSimilar(int4 scenePPF, int4 modelPPF) {
    if (scenePPF.x != modelPPF.x || scenePPF.y != modelPPF.y || scenePPF.z != modelPPF.z ||
        scenePPF.w != modelPPF.w)
        return false;
    return true;
}

__global__ void ppfKernel(float3 *points, float3 *norms, int4 *out, int count,
                           int refPointDownsampleFactor, float d_dist) {
/*
*这里的线层组织逻辑不好描述，建议按照流程一步一步在纸上走一遍便可以理解设计原因
*分块计算，提高处理效率。注意idx和各个索引(i,j)的计算
*/
    if(count <= 1) return;
    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;
    int bound;

    while(idx < count) {
        float3 thisPoint = points[idx];
        float3 thisNorm  = norms[idx];
        for(int i = 0; i < count; i+=BLOCK_SIZE){
            bound = MIN(count - i, BLOCK_SIZE);
            __syncthreads();
            for(int j = 0; j < bound; j++) {
                //this is to select the reference points,eg. refPointDownsampleFactor=1 means that:
                //every point need to calculate the PPF! Meanwhile refPointDownsampleFactor=5 means that:
                //every 5 points has one point to calculate the PPF
                if(idx % refPointDownsampleFactor != 0){
                    out[idx*count + j + i].x = HIP_NAN_F;
                    continue;
                } ;
                // handle case of identical points in pair
                if((j + i - idx) == 0){
                    out[idx*count + j + i].x = HIP_NAN_F;
                    continue;
                } ;
                float4 ppf = computePPF(thisPoint, thisNorm, points[i + j], norms[i + j], d_dist);

                out[idx*count + i + j] = discreteDisAndAngle(ppf, MIN_ANGLE, D_ANGLE);
            }
        }
        //grid stride
        __syncthreads();
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void ppfAngle(float3 *points, float3 *norms, float *out, int count,
                           int refPointDownsampleFactor, float d_dist){
    if(count <= 1)
        return;
    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;
    int bound;

    while(idx < count) {
        float3 thisPoint = points[idx];
        float3 thisNorm  = norms[idx];
        for(int i = 0; i < count; i+=BLOCK_SIZE){
            bound = MIN(count - i, BLOCK_SIZE);

            for(int j = 0; j < bound; j++) {
                if(idx % refPointDownsampleFactor != 0){
                    out[idx*count + j + i] = HIP_NAN_F;
                    continue;
                } ;
                if((j + i - idx) == 0){
                    out[idx*count + j + i] = HIP_NAN_F;
                    continue;
                } ;
                float alpha;
                transPointPair(thisPoint, thisNorm, points[i + j], alpha);
                out[idx*count + i + j] = alpha;
            }
        }
        __syncthreads();
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void ppf_hash_kernel(int4 *ppfs, unsigned int *codes, int count ) {
    if(count <= 1) return;

    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;

    while(idx < count) {
        if(ppfs[idx].x == HIP_NAN_F || ppfs[idx].x == -1) {
            codes[idx] = 0;
        } else {
            codes[idx] = hash(ppfs[idx]);
        }
        //grid stride
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void ppf_vote_count_kernel(unsigned int *sceneKeys, unsigned int *sceneIndices,
                                      unsigned int *hashKeys, std::size_t *ppfCount,
                                      unsigned long *ppf_vote_counts, int count){
    if(count <= 1) return;

    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;

    while(idx < count) {
        unsigned int thisSceneKey = sceneKeys[idx];
        unsigned int thisSceneIndex = sceneIndices[idx];
        if(thisSceneKey == 0 ||
           thisSceneKey != hashKeys[thisSceneIndex]) {
            ppf_vote_counts[idx] = 0;
        } else {
            ppf_vote_counts[idx] = ppfCount[thisSceneIndex];
        }
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void ppf_vote_kernel(unsigned int *sceneKeys, unsigned int *sceneIndices,
                                unsigned int *hashKeys, std::size_t *ppfCount,
                                std::size_t *firstPPFIndex, std::size_t *key2ppfMap,
                                float3 *modelPoints, float3 *modelNormals, int modelSize,
                                float3 *scenePoints, float3 *sceneNormals, int sceneSize,
                                int *voteAccumSpace, int4 *modelPPFs, int4 *scenePPFs,
                                float *modelAngles, float *sceneAngles, int count, float d_dist) {
    /*参数列表：
    * 1.scene所有点对hash值 2.scene所有点对与Hash值索引 
    * 3.unique hashes  4.hash counts 5.StartingIndices 6.hashToDataMap (参考论文27页和技术文档)
    * 7.model点  8.model点法向量  9.model点数量
    * 10.scene点 11.scene点法向量 12.scene点数量
    * 13.累加数组 14.model中所有点对PPFs 15.scene中所有点对PPFs
    * 16.model中所有点对的旋转角 17.scene中所有点对的旋转角
    * 18.scene中Hash值数量 19.d_dist(没用上)
    *每个thread完成对于一个点对的投票
    */
    if(count <= 1) return;

    int ind = threadIdx.x;
    int idx = ind + blockIdx.x * blockDim.x;
    unsigned int alpha_idx;
    float dis_thresh = 0;
    float angle_thresh = 0;

    while(idx < count) {
        unsigned int thisSceneKey = sceneKeys[idx];
        unsigned int thisSceneIndex = sceneIndices[idx];
        if (thisSceneKey == 0 ||
            thisSceneKey != hashKeys[thisSceneIndex]) {
            idx += blockDim.x * gridDim.x;
            continue;
        }
        unsigned int thisPPFCount = ppfCount[thisSceneIndex];
        unsigned int thisFirstPPFIndex = firstPPFIndex[thisSceneIndex];
        //get reference point ID
        unsigned int scene_r_index = idx / sceneSize;
        //get scene point ID
        unsigned int scene_i_index = idx - scene_r_index*sceneSize;

        int4 thisScenePPF = scenePPFs[scene_r_index * sceneSize + scene_i_index];

        if(thisScenePPF.x == -1) {
            continue;
        }
        float thisSceneAngle = sceneAngles[scene_r_index * sceneSize + scene_i_index];

        unsigned int modelPPFIndex, model_r_index, model_i_index;
        /*
        *以上都是计算索引
        *以下是对当前点对的Hash值，在Model Hash表中查找，找到相同Hash值的好多点对s，
        *依次比较，相同则投票(原子操作，不用结果也没变，用上保险)
        */
        for(int i = 0; i < thisPPFCount; i++) {
            modelPPFIndex = key2ppfMap[thisFirstPPFIndex+i];
            model_r_index = modelPPFIndex / modelSize;
            model_i_index = modelPPFIndex - model_r_index*modelSize;

            int4 thisModelPPF = modelPPFs[model_r_index * modelSize + model_i_index];
            float thisModelAngle = modelAngles[model_r_index * modelSize + model_i_index];

            if(isFeatureSimilar(thisScenePPF,thisModelPPF,dis_thresh,angle_thresh)) {
                transModelScene(thisModelAngle, thisSceneAngle, alpha_idx);

                atomicAdd(&voteAccumSpace[scene_r_index/5 * modelSize * N_ANGLE +
                                          model_r_index * N_ANGLE +
                                          alpha_idx],1);

            }
        }
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void addVote(int *d_accumSpace, int *d_votingPoint, int *d_votingNumber, int *d_votingAngle,
    int modelSize, int sceneSize, int *secondPoint, int *secondNumber, int *secondAngle){
    /*
    *对投票结束后的累加数组进行统计，找出每个参考点所对应数组中的
    *最大值和次大值(值代表：参考点所对应的model点，以及旋转角度，以及获票数)
    *kernel为最简单的2维thread排列
    */
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int rowLen = modelSize;
    int colLen = N_ANGLE;
    int maxAccum = -1;
    int idy_max, idx_max;
    int refPointNum = (sceneSize / 5) + 1;
    if(tid<refPointNum){
        for (int idy = 0; idy < rowLen; idy++) {
            for (int idx = 0; idx < colLen; idx++) {
                int votingValue = d_accumSpace[tid * modelSize * N_ANGLE +
                                               idy * N_ANGLE +
                                               idx];
                if (votingValue > maxAccum) {
                    maxAccum = votingValue;
                    idy_max = idy;
                    idx_max = idx;
                }
            }
        }
        d_votingPoint[tid] = idy_max;   //model point ID
        d_votingAngle[tid] = idx_max;   //rot angle
        d_votingNumber[tid] = maxAccum; //votes number

        d_accumSpace[tid * modelSize * N_ANGLE +
                    idy_max * N_ANGLE +
                    idx_max] = -1;
        idx_max = 0;
        idy_max = 0;
        maxAccum = -1;
        for (int idy = 0; idy < rowLen; idy++) {
            for (int idx = 0; idx < colLen; idx++) {
                int votingValue = d_accumSpace[tid * modelSize * N_ANGLE +
                                               idy * N_ANGLE +
                                               idx];
                if (votingValue > maxAccum) {
                    maxAccum = votingValue;
                    idy_max = idy;
                    idx_max = idx;
                }
            }
        }
        secondPoint[tid] = idy_max;   //model point ID
        secondAngle[tid] = idx_max;   //rot angle
        secondNumber[tid] = maxAccum; //votes number
    }
}

__host__ __device__ float dot(float3 v1, float3 v2){
    return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z;
}

__host__ __device__ float dot(float4 v1, float4 v2){
    return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z + v1.w*v2.w;
}

__host__ __device__ float norm(float3 v){
    return sqrtf(dot(v, v));
}

__host__ __device__ float norm(float4 v){
    return sqrtf(dot(v, v));
}

__host__ __device__ float3 normalize(float3 v){
    float length = sqrtf(dot(v, v));
    float3 w = {v.x / length, v.y / length, v.z / length};
    return w;
}

__host__ __device__ float3 cross(float3 v1, float3 v2){
    float3 w ={v1.y * v2.z - v1.z * v2.y,
               v1.z * v2.x - v1.x * v2.z,
               v1.x * v2.y - v1.y * v2.x};
    return w;
}

__host__ __device__ float3 operator*(float a, float3 v){
    float3 w = {a*v.x, a*v.y, a*v.z};
    return w;
}

__host__ __device__ float4 operator*(float a, float4 v){
    float4 w = {a*v.x, a*v.y, a*v.z, a*v.z};
    return w;
}

__host__ __device__ float3 operator+(float3 u, float3 v){
    float3 w = {u.x+v.x, u.y+v.y, u.z+v.z};
    return w;
}

__host__ __device__ float4 operator+(float4 u, float4 v){
    float4 w = {u.x+v.x, u.y+v.y, u.z+v.z, u.w+v.w};
    return w;
}

__host__ __device__ float3 operator-(float3 u, float3 v){
    float3 w = {u.x-v.x, u.y-v.y, u.z-v.z};
    return w;
}

__host__ __device__ float4 operator-(float4 u, float4 v){
    float4 w = {u.x-v.x, u.y-v.y, u.z-v.z, u.w-v.w};
    return w;
}


